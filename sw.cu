#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <bitset>
#include <iostream>
#include "cudadp.h"
#include "fasta_util.h"
using namespace std;

// Affine gap model
#define MATCH 1
#define MISMATCH 1
#define Gopen -3
#define Gext -2

#define M 100000
#define N 100000
#define G (1000*1000*1000)

struct Sequences {
    char *dev_A;
    char *dev_B;
};


__inline__ __device__
int3 cudadp_user_kernel(int i, int j, int3 left, int3 up, int3 diag, void* data) {

    struct Sequences* seq = (struct Sequences*)data;
    char *A = seq->dev_A;
    char *B = seq->dev_B;
    
    int3 result;
    result.x = max(left.x-Gext, left.z-Gopen);              // E[i,j]
    result.y = max(up.y-Gext, up.z-Gopen);                  // F[i,j]
    result.z = max(0, diag.z + (A[i]==B[j]?MATCH:MISMATCH));  // H[i,j]
    result.z = max3(result.z, result.x, result.y);          // H[i,j]

    return result;
}



string random_string(int length) {
    //srand (time(0) );
    string s(length, 'A');
    const char alphabet[] = {'A', 'C', 'G', 'T'};
    for(int i = 0; i < length; i++) {
        s[i] = alphabet[(rand() % 4)];
    }
    return s;
}


int main(int argc, char *argv[]) {
    //string A = read_fasta_file(argv[1]);
    //string B = read_fasta_file(argv[1]);
    //string A = "GTCTTACATCCGTTCG";
    //string B = "GTCTTACATCCGTTCG";
    string A = random_string(M);
    string B = random_string(N);
    //printf("A:%s\nB:%s\n", A.c_str(), B.c_str());

    struct Sequences seq;
    hipMalloc(&(seq.dev_A), sizeof(char) * A.length());
    hipMalloc(&(seq.dev_B), sizeof(char) * B.length());
    hipMemcpy(seq.dev_A, A.c_str(), sizeof(char)*A.length(), hipMemcpyHostToDevice);
    hipMemcpy(seq.dev_B, B.c_str(), sizeof(char)*B.length(), hipMemcpyHostToDevice);

    struct Sequences *dev_seq;
    hipMalloc(&dev_seq, sizeof(struct Sequences));
    hipMemcpy(dev_seq, &seq, sizeof(struct Sequences), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    DP_DiagUpLeft sw(40, 30);
    cudadp_start(&sw, dev_seq);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float gcpus = A.length() * B.length() * 1.0 / G / (milliseconds/1000.0);
    printf("time:%f, GCPUS: %f\n", milliseconds/1000.0, gcpus);


    hipFree(seq.dev_A);
    hipFree(seq.dev_B);
    hipFree(dev_seq);

    return 0;
}
